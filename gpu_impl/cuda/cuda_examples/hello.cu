
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void helloCUDA(float f)
{
    printf("Hello thread %d, f=%f\n", threadIdx.x, f);
}

int main()
{
    helloCUDA<<<1, 5>>>(1.2345f);
    hipDeviceSynchronize();
    return 0;
}
