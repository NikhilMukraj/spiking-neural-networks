#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define N 10


__global__ void vector_add(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i ++){
        out[i] = a[i] + b[i];
    }
}

int main(){
    float *a, *b, *out;
    float *d_a, *d_b, *d_out; 

    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    for(int i = 0; i < N; i++){
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    vector_add<<<5,2>>>(d_out, d_a, d_b, N);
    
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    for(int i = 0; i < N; i++){
        printf("%f\n", out[i]);
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    free(a); 
    free(b); 
    free(out);
}
